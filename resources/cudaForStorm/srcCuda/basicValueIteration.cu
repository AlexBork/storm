#include "hip/hip_runtime.h"
#include "basicValueIteration.h"

#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>
#include "hipsparse.h"

#include "cuspExtension.h"

__global__ void cuda_kernel_basicValueIteration_mvReduce(int const * const A, int * const B) {
	*B = *A;
}

template <typename IndexType, typename ValueType>
void basicValueIteration_mvReduce(uint_fast64_t const maxIterationCount, std::vector<IndexType> const& matrixRowIndices, std::vector<std::pair<IndexType, ValueType>> columnIndicesAndValues, std::vector<ValueType>& x, std::vector<ValueType> const& b, std::vector<IndexType> const& nondeterministicChoiceIndices) {
	IndexType* device_matrixRowIndices = nullptr;
	IndexType* device_matrixColIndicesAndValues = nullptr;
	ValueType* device_x = nullptr;
	ValueType* device_b = nullptr;
	ValueType* device_multiplyResult = nullptr;
	IndexType* device_nondeterministicChoiceIndices = nullptr;

	hipError_t cudaMallocResult;

	cudaMallocResult = hipMalloc<IndexType>(&device_matrixRowIndices, matrixRowIndices.size());
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Matrix Row Indices, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	cudaMallocResult = hipMalloc<IndexType>(&device_matrixColIndicesAndValues, columnIndicesAndValues.size() * 2);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Matrix Column Indices and Values, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	cudaMallocResult = hipMalloc<ValueType>(&device_x, x.size());
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector x, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	cudaMallocResult = hipMalloc<ValueType>(&device_b, b.size());
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector b, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	cudaMallocResult = hipMalloc<ValueType>(&device_multiplyResult, b.size());
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector multiplyResult, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	cudaMallocResult = hipMalloc<IndexType>(&device_nondeterministicChoiceIndices, nondeterministicChoiceIndices.size());
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Nondeterministic Choice Indices, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	// Memory allocated, copy data to device
	hipError_t cudaCopyResult;

	cudaCopyResult = hipMemcpy(device_matrixRowIndices, matrixRowIndices.data(), sizeof(IndexType) * matrixRowIndices.size(), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Matrix Row Indices, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	cudaCopyResult = hipMemcpy(device_matrixColIndicesAndValues, columnIndicesAndValues.data(), (sizeof(IndexType) * columnIndicesAndValues.size()) + (sizeof(ValueType) * columnIndicesAndValues.size()), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Matrix Column Indices and Values, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	cudaCopyResult = hipMemcpy(device_x, x.data(), sizeof(ValueType) * x.size(), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector x, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	cudaCopyResult = hipMemcpy(device_b, b.data(), sizeof(ValueType) * b.size(), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector b, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	cudaCopyResult = hipMemcpy(device_nondeterministicChoiceIndices, nondeterministicChoiceIndices.data(), sizeof(IndexType) * nondeterministicChoiceIndices.size(), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector b, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	// Data is on device, start Kernel


	// All code related to freeing memory and clearing up the device
cleanup:
	if (device_matrixRowIndices != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_matrixRowIndices);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Matrix Row Indices, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_matrixRowIndices = nullptr;
	}
	if (device_matrixColIndicesAndValues != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_matrixColIndicesAndValues);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Matrix Column Indices and Values, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_matrixColIndicesAndValues = nullptr;
	}
	if (device_x != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_x);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector x, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_x = nullptr;
	}
	if (device_b != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_b);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector b, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_b = nullptr;
	}
	if (device_multiplyResult != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_multiplyResult);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector multiplyResult, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_multiplyResult = nullptr;
	}
	if (device_nondeterministicChoiceIndices != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_nondeterministicChoiceIndices);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Nondeterministic Choice Indices, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_nondeterministicChoiceIndices = nullptr;
	}
}

/*
 * Declare and implement all exported functions for these Kernels here
 *
 */

void cudaForStormTestFunction(int a, int b) {
	std::cout << "Cuda for Storm: a + b = " << (a+b) << std::endl;
}

void basicValueIteration_mvReduce_uint64_double(uint_fast64_t const maxIterationCount, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<std::pair<uint_fast64_t, double>> columnIndicesAndValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices) {
	basicValueIteration_mvReduce<uint_fast64_t, double>(maxIterationCount, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices);
}