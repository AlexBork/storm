#include "hip/hip_runtime.h"
#include "basicValueIteration.h"

#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>
#include "hipsparse.h"


__global__ void cuda_kernel_basicValueIteration_mvReduce(int const * const A, int * const B) {
	*B = *A;
}

void cudaForStormTestFunction(int a, int b) {
	std::cout << "Cuda for Storm: a + b = " << (a+b) << std::endl;
}

void basicValueIteration_mvReduce(uint_fast64_t const maxIterationCount, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<std::pair<uint_fast64_t, double>> columnIndicesAndValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices) {
	if (sizeof(double) != sizeof(uint_fast64_t)) {
		std::cout << "FATAL ERROR - Internal Sizes of Double and uint_fast64_t do NOT match, CUDA acceleration not possible!" << std::endl;
		return;
	}
	
	uint_fast64_t* device_matrixRowIndices = nullptr;
	uint_fast64_t* device_matrixColIndicesAndValues = nullptr;
	double* device_x = nullptr;
	double* device_b = nullptr;
	double* device_multiplyResult = nullptr;
	uint_fast64_t* device_nondeterministicChoiceIndices = nullptr;

	hipError_t cudaMallocResult;

	cudaMallocResult = hipMalloc<uint_fast64_t>(&device_matrixRowIndices, matrixRowIndices.size());
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Matrix Row Indices, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	cudaMallocResult = hipMalloc<uint_fast64_t>(&device_matrixColIndicesAndValues, columnIndicesAndValues.size() * 2);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Matrix Column Indices and Values, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	cudaMallocResult = hipMalloc<double>(&device_x, x.size());
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector x, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	cudaMallocResult = hipMalloc<double>(&device_b, b.size());
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector b, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	cudaMallocResult = hipMalloc<double>(&device_multiplyResult, b.size());
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector multiplyResult, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	cudaMallocResult = hipMalloc<uint_fast64_t>(&device_nondeterministicChoiceIndices, nondeterministicChoiceIndices.size());
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Nondeterministic Choice Indices, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	// Memory allocated, copy data to device
	hipError_t cudaCopyResult;

	cudaCopyResult = hipMemcpy(device_matrixRowIndices, matrixRowIndices.data(), sizeof(uint_fast64_t) * matrixRowIndices.size(), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Matrix Row Indices, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	cudaCopyResult = hipMemcpy(device_matrixColIndicesAndValues, columnIndicesAndValues.data(), (sizeof(uint_fast64_t) * columnIndicesAndValues.size()) + (sizeof(double) * columnIndicesAndValues.size()), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Matrix Column Indices and Values, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	cudaCopyResult = hipMemcpy(device_x, x.data(), sizeof(double) * x.size(), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector x, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	cudaCopyResult = hipMemcpy(device_b, b.data(), sizeof(double) * b.size(), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector b, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	cudaCopyResult = hipMemcpy(device_nondeterministicChoiceIndices, nondeterministicChoiceIndices.data(), sizeof(uint_fast64_t) * nondeterministicChoiceIndices.size(), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector b, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	// Data is on device, start Kernel

	// All code related to freeing memory and clearing up the device
cleanup:
	if (device_matrixRowIndices != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_matrixRowIndices);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Matrix Row Indices, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_matrixRowIndices = nullptr;
	}
	if (device_matrixColIndicesAndValues != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_matrixColIndicesAndValues);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Matrix Column Indices and Values, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_matrixColIndicesAndValues = nullptr;
	}
	if (device_x != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_x);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector x, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_x = nullptr;
	}
	if (device_b != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_b);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector b, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_b = nullptr;
	}
	if (device_multiplyResult != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_multiplyResult);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector multiplyResult, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_multiplyResult = nullptr;
	}
	if (device_nondeterministicChoiceIndices != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_nondeterministicChoiceIndices);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Nondeterministic Choice Indices, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_nondeterministicChoiceIndices = nullptr;
	}
}

/*
void kernelSwitchTest(size_t N) {
	int* deviceIntA;
	int* deviceIntB;

	if (hipMalloc((void**)&deviceIntA, sizeof(int)) != hipSuccess) {
		std::cout << "Error in hipMalloc while allocating " << sizeof(int) << " Bytes!" << std::endl;
		return;
	}
	if (hipMalloc((void**)&deviceIntB, sizeof(int)) != hipSuccess) {
		std::cout << "Error in hipMalloc while allocating " << sizeof(int) << " Bytes!" << std::endl;
		return;
	}

	// Allocate space on the device
	auto start_time = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < N; ++i) {
		cuda_kernel_kernelSwitchTest<<<1,1>>>(deviceIntA, deviceIntB);
	}
	auto end_time = std::chrono::high_resolution_clock::now();
	std::cout << "Switching the Kernel " << N << " times took " << std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() << "micros" << std::endl;
	std::cout << "Resulting in " << (std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / ((double)(N))) << "Microseconds per Kernel Switch" << std::endl;

	// Free memory on device
	if (hipFree(deviceIntA) != hipSuccess) {
		std::cout << "Error in hipFree!" << std::endl;
		return;
	}
	if (hipFree(deviceIntB) != hipSuccess) {
		std::cout << "Error in hipFree!" << std::endl;
		return;
	}
}*/