#include "hip/hip_runtime.h"
#include "basicValueIteration.h"
#define CUSP_USE_TEXTURE_MEMORY

#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>
#include "hipsparse.h"

#include "utility.h"

#include "cuspExtension.h"
#include <thrust/transform.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>


#define CUDA_CHECK_ALL_ERRORS() do { \
	hipError_t errSync  = hipGetLastError(); \
	hipError_t errAsync = hipDeviceSynchronize(); \
	if (errSync != hipSuccess) { \
		std::cout << "(DLL) Sync kernel error: " << hipGetErrorString(errSync) << " (Code: " << errSync << ")" << std::endl; \
	} \
	if (errAsync != hipSuccess) { \
		std::cout << "(DLL) Async kernel error: " << hipGetErrorString(errAsync) << " (Code: " << errAsync << ")" << std::endl; \
	} } while(false)

__global__ void cuda_kernel_basicValueIteration_mvReduce(int const * const A, int * const B) {
	*B = *A;
}

template<typename T, bool Relative>
struct equalModuloPrecision : public thrust::binary_function<T,T,T>
{
__host__ __device__ T operator()(const T &x, const T &y) const
{
    if (Relative) {
		const T result = (x - y) / y;
		return (result > 0) ? result : -result;
    } else {
        const T result = (x - y);
		return (result > 0) ? result : -result;
    }
}
};

template <bool Minimize, bool Relative, typename IndexType, typename ValueType>
void basicValueIteration_mvReduce(uint_fast64_t const maxIterationCount, ValueType const precision, std::vector<IndexType> const& matrixRowIndices, std::vector<std::pair<IndexType, ValueType>> const& columnIndicesAndValues, std::vector<ValueType>& x, std::vector<ValueType> const& b, std::vector<IndexType> const& nondeterministicChoiceIndices) {
	IndexType* device_matrixRowIndices = nullptr;
	IndexType* device_matrixColIndicesAndValues = nullptr;
	ValueType* device_x = nullptr;
	ValueType* device_xSwap = nullptr;
	ValueType* device_b = nullptr;
	ValueType* device_multiplyResult = nullptr;
	IndexType* device_nondeterministicChoiceIndices = nullptr;

	std::cout.sync_with_stdio(true);
	std::cout << "(DLL) Device has " << getTotalCudaMemory() << " Bytes of Memory with " << getFreeCudaMemory() << "Bytes free (" << (static_cast<double>(getFreeCudaMemory()) / static_cast<double>(getTotalCudaMemory()))*100 << "%)." << std::endl; 
	size_t memSize = sizeof(IndexType) * matrixRowIndices.size() + sizeof(IndexType) * columnIndicesAndValues.size() * 2 + sizeof(ValueType) * x.size() + sizeof(ValueType) * x.size() + sizeof(ValueType) * b.size() + sizeof(ValueType) * b.size() + sizeof(IndexType) * nondeterministicChoiceIndices.size();
	std::cout << "(DLL) We will allocate " << memSize << " Bytes." << std::endl;

	const IndexType matrixRowCount = matrixRowIndices.size() - 1;
	const IndexType matrixColCount = nondeterministicChoiceIndices.size() - 1;
	const IndexType matrixNnzCount = columnIndicesAndValues.size();

	hipError_t cudaMallocResult;

	bool converged = false;
	uint_fast64_t iterationCount = 0;

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_matrixRowIndices), sizeof(IndexType) * (matrixRowCount + 1));
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Matrix Row Indices, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_matrixColIndicesAndValues), sizeof(IndexType) * matrixNnzCount + sizeof(ValueType) * matrixNnzCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Matrix Column Indices and Values, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_x), sizeof(ValueType) * matrixColCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector x, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_xSwap), sizeof(ValueType) * matrixColCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector x swap, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_b), sizeof(ValueType) * matrixRowCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector b, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_multiplyResult), sizeof(ValueType) * matrixRowCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector multiplyResult, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_nondeterministicChoiceIndices), sizeof(IndexType) * (matrixRowCount + 1));
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Nondeterministic Choice Indices, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	// Memory allocated, copy data to device
	hipError_t cudaCopyResult;

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_matrixRowIndices, matrixRowIndices.data(), sizeof(IndexType) * (matrixRowCount + 1), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Matrix Row Indices, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_matrixColIndicesAndValues, columnIndicesAndValues.data(), (sizeof(IndexType) * matrixNnzCount) + (sizeof(ValueType) * matrixNnzCount), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Matrix Column Indices and Values, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_x, x.data(), sizeof(ValueType) * matrixColCount, hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector x, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	// Preset the xSwap to zeros...
	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemset(device_xSwap, 0, sizeof(ValueType) * matrixColCount);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not zero the Swap Vector x, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_b, b.data(), sizeof(ValueType) * matrixRowCount, hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector b, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	// Preset the multiplyResult to zeros...
	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemset(device_multiplyResult, 0, sizeof(ValueType) * matrixRowCount);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not zero the multiply Result, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_nondeterministicChoiceIndices, nondeterministicChoiceIndices.data(), sizeof(IndexType) * (matrixRowCount + 1), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector b, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	// Data is on device, start Kernel
	while (!converged && iterationCount < maxIterationCount)
	{ // In a sub-area since transfer of control via label evades initialization
		cusp::detail::device::storm_cuda_opt_spmv_csr_vector<IndexType, ValueType>(matrixRowCount, matrixNnzCount, device_matrixRowIndices, device_matrixColIndicesAndValues, device_x, device_multiplyResult);
		CUDA_CHECK_ALL_ERRORS();

		thrust::device_ptr<ValueType> devicePtrThrust_b(device_b);
		thrust::device_ptr<ValueType> devicePtrThrust_multiplyResult(device_multiplyResult);

		// Transform: Add multiplyResult + b inplace to multiplyResult
		thrust::transform(devicePtrThrust_multiplyResult, devicePtrThrust_multiplyResult + matrixRowCount, devicePtrThrust_b, devicePtrThrust_multiplyResult, thrust::plus<ValueType>());
		CUDA_CHECK_ALL_ERRORS();

		// Reduce: Reduce multiplyResult to a new x vector
		cusp::detail::device::storm_cuda_opt_vector_reduce<Minimize, IndexType, ValueType>(matrixColCount, matrixRowCount, device_nondeterministicChoiceIndices, device_xSwap, device_multiplyResult);
		CUDA_CHECK_ALL_ERRORS();

		// Check for convergence
		// Transform: x = abs(x - xSwap)/ xSwap
		thrust::device_ptr<ValueType> devicePtrThrust_x(device_x);
		thrust::device_ptr<ValueType> devicePtrThrust_x_end(device_x + matrixColCount);
		thrust::device_ptr<ValueType> devicePtrThrust_xSwap(device_xSwap);
		thrust::transform(devicePtrThrust_x, devicePtrThrust_x_end, devicePtrThrust_xSwap, devicePtrThrust_x, equalModuloPrecision<ValueType, Relative>());
		CUDA_CHECK_ALL_ERRORS();

		// Reduce: get Max over x and check for res < Precision
		ValueType maxX = thrust::reduce(devicePtrThrust_x, devicePtrThrust_x_end, 0, thrust::maximum<ValueType>());
		CUDA_CHECK_ALL_ERRORS();
		converged = maxX < precision;
		++iterationCount;

		// Swap pointers, device_x always contains the most current result
		std::swap(device_x, device_xSwap);
	}
	std::cout << "(DLL) Executed " << iterationCount << " of max. " << maxIterationCount << " Iterations." << std::endl;

	// Get x back from the device
	cudaCopyResult = hipMemcpy(x.data(), device_x, sizeof(ValueType) * matrixColCount, hipMemcpyDeviceToHost);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy back data for result vector x, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	// All code related to freeing memory and clearing up the device
cleanup:
	if (device_matrixRowIndices != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_matrixRowIndices);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Matrix Row Indices, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_matrixRowIndices = nullptr;
	}
	if (device_matrixColIndicesAndValues != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_matrixColIndicesAndValues);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Matrix Column Indices and Values, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_matrixColIndicesAndValues = nullptr;
	}
	if (device_x != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_x);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector x, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_x = nullptr;
	}
	if (device_xSwap != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_xSwap);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector x swap, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_xSwap = nullptr;
	}
	if (device_b != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_b);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector b, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_b = nullptr;
	}
	if (device_multiplyResult != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_multiplyResult);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector multiplyResult, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_multiplyResult = nullptr;
	}
	if (device_nondeterministicChoiceIndices != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_nondeterministicChoiceIndices);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Nondeterministic Choice Indices, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_nondeterministicChoiceIndices = nullptr;
	}
}

template <typename IndexType, typename ValueType>
void basicValueIteration_spmv(uint_fast64_t const matrixColCount, std::vector<IndexType> const& matrixRowIndices, std::vector<std::pair<IndexType, ValueType>> const& columnIndicesAndValues, std::vector<ValueType> const& x, std::vector<ValueType>& b) {
	IndexType* device_matrixRowIndices = nullptr;
	IndexType* device_matrixColIndicesAndValues = nullptr;
	ValueType* device_x = nullptr;
	ValueType* device_multiplyResult = nullptr;

	std::cout.sync_with_stdio(true);
	std::cout << "(DLL) Device has " << getTotalCudaMemory() << " Bytes of Memory with " << getFreeCudaMemory() << "Bytes free (" << (static_cast<double>(getFreeCudaMemory()) / static_cast<double>(getTotalCudaMemory()))*100 << "%)." << std::endl; 
	size_t memSize = sizeof(IndexType) * matrixRowIndices.size() + sizeof(IndexType) * columnIndicesAndValues.size() * 2 + sizeof(ValueType) * x.size() + sizeof(ValueType) * b.size();
	std::cout << "(DLL) We will allocate " << memSize << " Bytes." << std::endl;

	const IndexType matrixRowCount = matrixRowIndices.size() - 1;
	const IndexType matrixNnzCount = columnIndicesAndValues.size();

	hipError_t cudaMallocResult;

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_matrixRowIndices), sizeof(IndexType) * (matrixRowCount + 1));
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Matrix Row Indices, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_matrixColIndicesAndValues), sizeof(IndexType) * matrixNnzCount + sizeof(ValueType) * matrixNnzCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Matrix Column Indices and Values, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_x), sizeof(ValueType) * matrixColCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector x, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaMallocResult = hipMalloc(reinterpret_cast<void**>(&device_multiplyResult), sizeof(ValueType) * matrixRowCount);
	if (cudaMallocResult != hipSuccess) {
		std::cout << "Could not allocate memory for Vector multiplyResult, Error Code " << cudaMallocResult << "." << std::endl;
		goto cleanup;
	}

	// Memory allocated, copy data to device
	hipError_t cudaCopyResult;

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_matrixRowIndices, matrixRowIndices.data(), sizeof(IndexType) * (matrixRowCount + 1), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Matrix Row Indices, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_matrixColIndicesAndValues, columnIndicesAndValues.data(), (sizeof(IndexType) * matrixNnzCount) + (sizeof(ValueType) * matrixNnzCount), hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Matrix Column Indices and Values, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemcpy(device_x, x.data(), sizeof(ValueType) * matrixColCount, hipMemcpyHostToDevice);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy data for Vector x, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	// Preset the multiplyResult to zeros...
	CUDA_CHECK_ALL_ERRORS();
	cudaCopyResult = hipMemset(device_multiplyResult, 0, sizeof(ValueType) * matrixRowCount);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not zero the multiply Result, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	cusp::detail::device::storm_cuda_opt_spmv_csr_vector<IndexType, ValueType>(matrixRowCount, matrixNnzCount, device_matrixRowIndices, device_matrixColIndicesAndValues, device_x, device_multiplyResult);
	CUDA_CHECK_ALL_ERRORS();

	// Get result back from the device
	cudaCopyResult = hipMemcpy(b.data(), device_multiplyResult, sizeof(ValueType) * matrixRowCount, hipMemcpyDeviceToHost);
	if (cudaCopyResult != hipSuccess) {
		std::cout << "Could not copy back data for result vector, Error Code " << cudaCopyResult << std::endl;
		goto cleanup;
	}

	// All code related to freeing memory and clearing up the device
cleanup:
	if (device_matrixRowIndices != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_matrixRowIndices);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Matrix Row Indices, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_matrixRowIndices = nullptr;
	}
	if (device_matrixColIndicesAndValues != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_matrixColIndicesAndValues);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Matrix Column Indices and Values, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_matrixColIndicesAndValues = nullptr;
	}
	if (device_x != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_x);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector x, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_x = nullptr;
	}
	if (device_multiplyResult != nullptr) {
		hipError_t cudaFreeResult = hipFree(device_multiplyResult);
		if (cudaFreeResult != hipSuccess) {
			std::cout << "Could not free Memory of Vector multiplyResult, Error Code " << cudaFreeResult << "." << std::endl;
		}
		device_multiplyResult = nullptr;
	}
}

/*
 * Declare and implement all exported functions for these Kernels here
 *
 */

void cudaForStormTestFunction(int a, int b) {
	std::cout << "Cuda for Storm: a + b = " << (a+b) << std::endl;
}

void basicValueIteration_spmv_uint64_double(uint_fast64_t const matrixColCount, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<std::pair<uint_fast64_t, double>> const& columnIndicesAndValues, std::vector<double> const& x, std::vector<double>& b) {
	basicValueIteration_spmv(matrixColCount, matrixRowIndices, columnIndicesAndValues, x, b);
}

void basicValueIteration_mvReduce_uint64_double_minimize(uint_fast64_t const maxIterationCount, double const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<std::pair<uint_fast64_t, double>> const& columnIndicesAndValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices) {
	if (relativePrecisionCheck) {
		basicValueIteration_mvReduce<true, true, uint_fast64_t, double>(maxIterationCount, precision, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices);
	} else {
		basicValueIteration_mvReduce<true, false, uint_fast64_t, double>(maxIterationCount, precision, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices);
	}
}

void basicValueIteration_mvReduce_uint64_double_maximize(uint_fast64_t const maxIterationCount, double const precision, bool const relativePrecisionCheck, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<std::pair<uint_fast64_t, double>> const& columnIndicesAndValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices) {
	if (relativePrecisionCheck) {
		basicValueIteration_mvReduce<false, true, uint_fast64_t, double>(maxIterationCount, precision, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices);
	} else {
		basicValueIteration_mvReduce<false, false, uint_fast64_t, double>(maxIterationCount, precision, matrixRowIndices, columnIndicesAndValues, x, b, nondeterministicChoiceIndices);
	}
}