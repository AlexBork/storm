#include "hip/hip_runtime.h"
#include "basicValueIteration.h"

#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>
#include "hipsparse.h"


__global__ void cuda_kernel_basicValueIteration_mvReduce(int const * const A, int * const B) {
	*B = *A;
}

void basicValueIteration_mvReduce(uint_fast64_t const maxIterationCount, std::vector<uint_fast64_t> const& matrixRowIndices, std::vector<uint_fast64_t> const& matrixColumnIndices, std::vector<double> const& matrixValues, std::vector<double>& x, std::vector<double> const& b, std::vector<uint_fast64_t> const& nondeterministicChoiceIndices) {
	std::cout << "basicValueIteration_mvReduce is implemented for ValueType == double :)" << std::endl;
}

/*
void kernelSwitchTest(size_t N) {
	int* deviceIntA;
	int* deviceIntB;

	if (hipMalloc((void**)&deviceIntA, sizeof(int)) != hipSuccess) {
		std::cout << "Error in hipMalloc while allocating " << sizeof(int) << " Bytes!" << std::endl;
		return;
	}
	if (hipMalloc((void**)&deviceIntB, sizeof(int)) != hipSuccess) {
		std::cout << "Error in hipMalloc while allocating " << sizeof(int) << " Bytes!" << std::endl;
		return;
	}

	// Allocate space on the device
	auto start_time = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < N; ++i) {
		cuda_kernel_kernelSwitchTest<<<1,1>>>(deviceIntA, deviceIntB);
	}
	auto end_time = std::chrono::high_resolution_clock::now();
	std::cout << "Switching the Kernel " << N << " times took " << std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() << "micros" << std::endl;
	std::cout << "Resulting in " << (std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / ((double)(N))) << "Microseconds per Kernel Switch" << std::endl;

	// Free memory on device
	if (hipFree(deviceIntA) != hipSuccess) {
		std::cout << "Error in hipFree!" << std::endl;
		return;
	}
	if (hipFree(deviceIntB) != hipSuccess) {
		std::cout << "Error in hipFree!" << std::endl;
		return;
	}
}*/