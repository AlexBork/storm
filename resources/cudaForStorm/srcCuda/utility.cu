
#include <hip/hip_runtime.h>
size_t getFreeCudaMemory() {
	size_t freeMemory;
	size_t totalMemory;
	hipMemGetInfo(&freeMemory, &totalMemory);

	return freeMemory;
}

size_t getTotalCudaMemory() {
	size_t freeMemory;
	size_t totalMemory;
	hipMemGetInfo(&freeMemory, &totalMemory);

	return totalMemory;
}

void resetCudaDevice() {
	hipDeviceReset();
}